#include "hip/hip_runtime.h"
/**
 * @file cleaver_cuda.cu
 * This file contains the GPU calls for Cleaver.
 * @version April 21, 2014
 * @author: Brig Bagley
 */
#include <cstdio>
#include <iostream>
#include <stdint.h>
#include "cleaver_cuda.hh"

namespace CleaverCUDA {
/**
 * The error checker for CUDA.
 * @param val The return value from a CUDA Call.
 */
void CudaCheckReturn(hipError_t val) {
  if (val != hipSuccess) {
    fprintf(stderr, "Error %s at line %d in file %s\n",
            hipGetErrorString(val), __LINE__, __FILE__);
    exit(1);
  }
}

__device__ __host__
float DataTransformCUDA(float *data_,
                        float i, float j, float k,
                        size_t mm, size_t m, float* scales,
                        float* scale,
                        size_t ww_, size_t hh_, size_t dd_) {

  float x = i;
  float y = j;
  float z = k;

  size_t whd = ww_*hh_*dd_;
  size_t wh = ww_*hh_;
  size_t w = ww_;
  size_t h = hh_;
  size_t d = dd_;

  x *= scale[0];
  y *= scale[1];
  z *= scale[2];
  if (mm < m - 1) {
    x *= scales[mm*3+0];
    y *= scales[mm*3+1];
    z *= scales[mm*3+2];
  }

  x -= 0.5f;
  y -= 0.5f;
  z -= 0.5f;

  bool inside = (x >= -.5) && (x < ww_+.5) &&
      (y >= -.5) && (y < hh_+.5) &&
      (z >= -.5) && (z < dd_+.5);

  if (mm < m - 1) {
    if(inside) {
      float t = fmodf(x,1.0f);
      float u = fmodf(y,1.0f);
      float v = fmodf(z,1.0f);

      int i0 = (int)(floorf(x));   int i1 = i0+1;
      int j0 = (int)(floorf(y));   int j1 = j0+1;
      int k0 = (int)(floorf(z));   int k1 = k0+1;
      int zero = 0;

      i0 = min(max(zero,i0), (int)(w)-1);
      j0 = min(max(zero,j0),(int)(h)-1);
      k0 = min(max(zero,k0),(int)(d)-1);

      i1 = min(max(zero,i1),(int)(w)-1);
      j1 = min(max(zero,j1),(int)(h)-1);
      k1 = min(max(zero,k1),(int)(d)-1);

      float C000 = data_[i0 + j0*w + k0*wh + mm*whd];
      float C001 = data_[i0 + j0*w + k1*wh + mm*whd];
      float C010 = data_[i0 + j1*w + k0*wh + mm*whd];
      float C011 = data_[i0 + j1*w + k1*wh + mm*whd];
      float C100 = data_[i1 + j0*w + k0*wh + mm*whd];
      float C101 = data_[i1 + j0*w + k1*wh + mm*whd];
      float C110 = data_[i1 + j1*w + k0*wh + mm*whd];
      float C111 = data_[i1 + j1*w + k1*wh + mm*whd];

      return float((1-t)*(1-u)*(1-v)*C000 + (1-t)*(1-u)*(v)*C001 +
                   (1-t)*  (u)*(1-v)*C010 + (1-t)*  (u)*(v)*C011 +
                   (t)*(1-u)*(1-v)*C100 +   (t)*(1-u)*(v)*C101 +
                   (t)*  (u)*(1-v)*C110 +   (t)*  (u)*(v)*C111);
    } else
      return -1000.;
  } else {
    if(inside)
      return -1000.;
    else
      return 1000.;
  }

}

/**
 * Runs the loops of chunks to find max materials.
 * @param input_device_memory Pointer to the material data.
 * @param w The input data width.
 * @param h The input data height.
 * @param d The input data depth.
 * @param m The input data number of materials.
 * @param output_device_memory The pointer to the chunk of results.
 * @param i The x location of the start of the chunk.
 * @param j The y location of the start of the chunk.
 * @param k The z location of the start of the chunk.
 * @param endi The x location of the end of the chunk.
 * @param endj The y location of the end of the chunk.
 * @param endk The z location of the end of the chunk.
 * @param device_scales The pointer to the scales to use.
 * @param device_scale The pointer to the scale to use.
 */
__global__
void FindMaxesCUDA(void *input_device_memory,
                   size_t w, size_t h, size_t d, size_t m,
                   void *output_device_memory,
                   size_t i, size_t j, size_t k,
                   size_t endi, size_t endj, size_t endk,
                   void *device_scales, void * device_scale) {
  //cast to proper data
  float* data_ = (float*)input_device_memory;
  float* scales = (float*)device_scales;
  float* scale = (float*)device_scale;
  char* labels = (char*)output_device_memory;

  size_t grid_max = gridDim.x * gridDim.y * gridDim.z;

  size_t idx =
      blockIdx.x       +
      blockIdx.y * gridDim.x  +
      blockIdx.z * gridDim.x * gridDim.y  +
      threadIdx.x  * grid_max  +
      threadIdx.y  * grid_max * blockDim.x +
      threadIdx.z  * grid_max * blockDim.x * blockDim.y;

  size_t x = idx % kChunkSize;
  idx -= x; idx /= kChunkSize;
  size_t y = idx % kChunkSize;
  idx -= y; idx /= kChunkSize;
  size_t z = idx % kChunkSize;

  //  for (size_t x = 0; x < (kBlockSize); x++ )
  //    for (size_t y = 0; y < (kBlockSize); y++ )
  //      for (size_t z = 0; z < (kBlockSize); z++ )
  if ((x < endi - i) && (y < endj - j) && (z < endk - k))
    labels[x +
           y * kChunkSize +
           z * kChunkSize * kChunkSize] =
               GetLabelCUDA(data_,scales, scale,i+x,j+y,k+z,m,w,h,d);
}

__device__ __host__
float GetCellCenterValueCUDA(
    float* data_, float* scales, float* scale,
    size_t i, size_t j, size_t k,
    size_t m, size_t num_mats,
    bool find_max, char* max_mat,
    size_t w, size_t h, size_t d) {
  if (!find_max) {
    *max_mat = m;
    return DataTransformCUDA(
        data_,
        static_cast<float>(i)+0.5f,
        static_cast<float>(j)+0.5f,
        static_cast<float>(k)+0.5f,
        m,num_mats,scales,scale,w,h,d);
  }
  char mat = 0;
  float tmp, val = DataTransformCUDA(
      data_,
      static_cast<float>(i)+0.5f,
      static_cast<float>(j)+0.5f,
      static_cast<float>(k)+0.5f,
      0,num_mats,scales,scale,w,h,d);
  for(size_t a = 1; a < num_mats; a++) {
    if ((tmp = DataTransformCUDA(
        data_,
        static_cast<float>(i)+0.5f,
        static_cast<float>(j)+0.5f,
        static_cast<float>(k)+0.5f,
        a,num_mats,scales,scale,w,h,d)) > val) {
      val = tmp;
      mat = a;
    }
  }
  *max_mat = mat;
  return val;
}

__device__ __host__
void SetArrayCUDA(size_t *cell, size_t i, size_t j, size_t k) {
  cell[0] = i;
  cell[1] = j;
  cell[2] = k;
}

__device__ __host__
void GetAdjacentCellFromEdgeCUDA(
    CleaverCUDA::edge_index num, bool first,
    size_t i, size_t j, size_t k, size_t* cell) {
  switch (num) {
    //Diagonal edges
    case CleaverCUDA::DULF:
      SetArrayCUDA(cell,i,j+1,k); return;
    case CleaverCUDA::DULB:
      SetArrayCUDA(cell,i,j+1,k+1); return;
    case CleaverCUDA::DURF:
      SetArrayCUDA(cell,i+1,j+1,k); return;
    case CleaverCUDA::DURB:
      SetArrayCUDA(cell,i+1,j+1,k+1); return;
    case CleaverCUDA::DLLF:
      SetArrayCUDA(cell,i,j,k); return;
    case CleaverCUDA::DLLB:
      SetArrayCUDA(cell,i,j,k+1); return;
    case CleaverCUDA::DLRF:
      SetArrayCUDA(cell,i+1,j,k); return;
    case CleaverCUDA::DLRB:
      SetArrayCUDA(cell,i+1,j,k+1); return;
      //Dual Edges
    case CleaverCUDA::CL:
      SetArrayCUDA(cell,i-1,j,k); return;
    case CleaverCUDA::CR:
      SetArrayCUDA(cell,i+1,j,k); return;
    case CleaverCUDA::CU:
      SetArrayCUDA(cell,i,j+1,k); return;
    case CleaverCUDA::CD:
      SetArrayCUDA(cell,i,j-1,k); return;
    case CleaverCUDA::CF:
      SetArrayCUDA(cell,i,j,k-1); return;
    case CleaverCUDA::CB:
      SetArrayCUDA(cell,i,j,k+1); return;
      //Axis edges (top)
    case CleaverCUDA::UL:
      if (first) SetArrayCUDA(cell,i,j+1,k);
      else
        SetArrayCUDA(cell,i,j+1,k+1);
      return;
    case CleaverCUDA::UR:
      if (first) SetArrayCUDA(cell, i+1,j+1,k);
      else
        SetArrayCUDA(cell,i+1,j+1,k+1);
      return;
    case CleaverCUDA::UF:
      if (first) SetArrayCUDA(cell,i,j+1,k);
      else
        SetArrayCUDA(cell,i+1,j+1,k);
      return;
    case CleaverCUDA::UB:
      if (first) SetArrayCUDA(cell,i,j+1,k+1);
      else
        SetArrayCUDA(cell,i+1,j+1,k+1);
      return;
      //axis edges (bottom)
    case CleaverCUDA::LL:
      if (first) SetArrayCUDA(cell,i,j,k);
      else
        SetArrayCUDA(cell,i,j,k+1);
      return;
    case CleaverCUDA::LR:
      if (first) SetArrayCUDA(cell,i+1,j,k);
      else
        SetArrayCUDA(cell,i+1,j,k+1);
      return;
    case CleaverCUDA::LF:
      if (first) SetArrayCUDA(cell,i,j,k);
      else
        SetArrayCUDA(cell,i+1,j,k);
      return;
    case CleaverCUDA::LB:
      if (first) SetArrayCUDA(cell,i,j,k+1);
      else
        SetArrayCUDA(cell,i+1,j,k+1);
      return;
      //axis edges (columns)
    case CleaverCUDA::FL:
      if (first) SetArrayCUDA(cell,i,j,k);
      else
        SetArrayCUDA(cell,i,j+1,k);
      return;
    case CleaverCUDA::FR:
      if (first) SetArrayCUDA(cell,i+1,j,k);
      else
        SetArrayCUDA(cell,i+1,j+1,k);
      return;
    case CleaverCUDA::BL:
      if (first) SetArrayCUDA(cell,i,j,k+1);
      else
        SetArrayCUDA(cell,i,j+1,k+1);
      return;
    case CleaverCUDA::BR:
      if (first) SetArrayCUDA(cell,i+1,j,k+1);
      else
        SetArrayCUDA(cell,i+1,j+1,k+1);
      return;
  }
}

__device__ __host__
char GetLabelCUDA(float* data_, float* scales, float* scale,
              size_t i, size_t j, size_t k, char num_mats,
              size_t w, size_t h, size_t d) {
  float max = DataTransformCUDA(data_, static_cast<float>(i),
                                static_cast<float>(j),
                                static_cast<float>(k),
                                0,num_mats,scales,scale,w,h,d);
  char max_mat = 0;
  for(char t = 1; t < num_mats; t++) {
    float tmp;
    if ((tmp = DataTransformCUDA(data_, static_cast<float>(i),
                                 static_cast<float>(j),
                                 static_cast<float>(k),
                                 t,num_mats,scales,scale,w,h,d)) > max) {
      max = tmp;
      max_mat = t;
    }
  }
  return max_mat;
}

__device__ __host__
float GetEdgeMatAndValueAtEndpointCUDA(
    float* data_, float* scales, float* scale,
    CleaverCUDA::edge_index num, bool first, bool find_max,
    size_t m, size_t num_mats, char* ret_mat,
    size_t i, size_t j, size_t k,
    size_t w, size_t h, size_t d) {
  //find the respective adjacent cell for the vertex we want
  size_t arr[3];
  CleaverCUDA::GetAdjacentCellFromEdgeCUDA(num,first,i,j,k,arr);
  //the center value of this cell. (V1)
  char mat;
  float res = CleaverCUDA::GetCellCenterValueCUDA(
      data_,scales,scale,i,j,k,
      m,num_mats,find_max,&mat,w,h,d);
  if ((num < 8 && !first) || (num >= 14)) {
    // diagonal edges & second vertex, or axis edges.
    mat = find_max?CleaverCUDA::GetLabelCUDA(
        data_,scales,scale,arr[0],arr[1],arr[2],
        num_mats,w,h,d):m;
    res = CleaverCUDA::DataTransformCUDA(
        data_,arr[0],arr[1],arr[2],mat,num_mats,
        scales,scale,w,h,d);
  } else if ((8 <= num && num < 14)) { // dual edges
    bool neg =
        (num == CleaverCUDA::CL) || // is always on left, bottom, or front.
        (num == CleaverCUDA::CD) ||
        (num == CleaverCUDA::CF);
    if (neg) first = !first;       // flip first if we are on the negative edge
    if (!first)                    // second vertex
      res = CleaverCUDA::GetCellCenterValueCUDA(
          data_,scales,scale,arr[0],arr[1],arr[2],
          m,num_mats,find_max,&mat,w,h,d);
  }
  *ret_mat = mat;
  return res;
}

__device__ __host__
void GetEdgeVerticesCUDA(
    CleaverCUDA::edge_index num,
    size_t i, size_t j, size_t k,
    float* scale, float verts[2][3]) {
  float x = static_cast<float>(i) * scale[0];
  float y = static_cast<float>(j) * scale[1];
  float z = static_cast<float>(k) * scale[2];
  float v1[3] = {x,y,z};
  float v2[3] = {x+scale[0],y,z};
  float v3[3] = {x,y+scale[1],z};
  float v4[3] = {x+scale[0],y+scale[1],z};
  float v5[3] = {x,y,z+scale[2]};
  float v6[3] = {x+scale[0],y,z+scale[2]};
  float v7[3] = {x,y+scale[1],z+scale[2]};
  float v8[3] = {x+scale[0],y+scale[1],z+scale[2]};
  float v9[3] = {x+.5f * scale[0],
      y+.5f * scale[1],z+.5f * scale[2]};
  float v10[3] = {x-.5f * scale[0],
      y+.5f * scale[1],z+.5f * scale[2]};
  float v11[3] = {x+1.5f * scale[0],
      y+.5f * scale[1],z+.5f * scale[2]};
  float v12[3] = {x+.5f * scale[0],
      y-.5f * scale[1],z+.5f * scale[2]};
  float v13[3] = {x+.5f * scale[0],
      y+1.5f * scale[1],z+.5f * scale[2]};
  float v14[3] = {x+.5f * scale[0],
      y+.5f * scale[1],z-.5f * scale[2]};
  float v15[3] = {x+.5f * scale[0],
      y+.5f * scale[1],z+1.5f * scale[2]};

  float* ans[2] = {NULL, NULL};
  switch (num) {
    //diagonal edges upper
    case CleaverCUDA::DULF: ans[0] = v9; ans[1] = v3; break;
    case CleaverCUDA::DULB: ans[0] = v9; ans[1] = v7; break;
    case CleaverCUDA::DURF: ans[0] = v9; ans[1] = v4; break;
    case CleaverCUDA::DURB: ans[0] = v9; ans[1] = v8; break;
    //diagonal edges lower
    case CleaverCUDA::DLLF: ans[0] = v9; ans[1] = v1; break;
    case CleaverCUDA::DLLB: ans[0] = v9; ans[1] = v5; break;
    case CleaverCUDA::DLRF: ans[0] = v9; ans[1] = v2; break;
    case CleaverCUDA::DLRB: ans[0] = v9; ans[1] = v6; break;
    //dual edges
    case CleaverCUDA::CL: ans[0] = v10; ans[1] = v9; break;
    case CleaverCUDA::CR: ans[0] = v9; ans[1] = v11; break;
    case CleaverCUDA::CU: ans[0] = v9; ans[1] = v13; break;
    case CleaverCUDA::CD: ans[0] = v12; ans[1] = v9; break;
    case CleaverCUDA::CF: ans[0] = v14; ans[1] = v9; break;
    case CleaverCUDA::CB: ans[0] = v9; ans[1] = v15; break;
    //top face edges
    case CleaverCUDA::UL: ans[0] = v3; ans[1] = v7; break;
    case CleaverCUDA::UR: ans[0] = v4; ans[1] = v8; break;
    case CleaverCUDA::UF: ans[0] = v3; ans[1] = v4; break;
    case CleaverCUDA::UB: ans[0] = v7; ans[1] = v8; break;
    //bottom face edges
    case CleaverCUDA::LL: ans[0] = v1; ans[1] = v5; break;
    case CleaverCUDA::LR: ans[0] = v2; ans[1] = v6; break;
    case CleaverCUDA::LF: ans[0] = v1; ans[1] = v2; break;
    case CleaverCUDA::LB: ans[0] = v5; ans[1] = v6; break;
    //column edges
    case CleaverCUDA::FL: ans[0] = v1; ans[1] = v3; break;
    case CleaverCUDA::FR: ans[0] = v2; ans[1] = v4; break;
    case CleaverCUDA::BL: ans[0] = v5; ans[1] = v7; break;
    case CleaverCUDA::BR: ans[0] = v6; ans[1] = v8; break;
  }
  for (size_t t = 0; t < 2; t++)
    for (size_t tt = 0; tt < 3; tt++)
      verts[t][tt] = ans[t][tt];
}

__device__ __host__
void FindEdgeCutCUDA(
    float* data_,
    float* scales,
    float* scale,
    size_t w, size_t h, size_t d, size_t m,
    size_t i, size_t j, size_t k,
    Edge* edge, CleaverCUDA::edge_index num) {
  edge->isCut_eval |= CleaverCUDA::kIsEvaluated;
  //get strongest material at each end of the edge
  unsigned char matA, matB, dummy;
  float v1 = CleaverCUDA::GetEdgeMatAndValueAtEndpointCUDA(
      data_,scales,scale,num,true,true,
      0,m,(char*)&matA,i,j,k,w,h,d);
  float v2 = CleaverCUDA::GetEdgeMatAndValueAtEndpointCUDA(
      data_,scales,scale,num,false,true,
      0,m,(char*)&matB,i,j,k,w,h,d);
  //if they are the same, nothing to be done: no cut
  if (matA == matB) return;
  //if they are different, interpolate transition point.
  float a1 = v1;
  float b2 = v2;
  float a2 = CleaverCUDA::GetEdgeMatAndValueAtEndpointCUDA(
      data_,scales,scale,num,false,false,
      matA,m,(char*)&dummy,i,j,k,w,h,d);
  float b1 = CleaverCUDA::GetEdgeMatAndValueAtEndpointCUDA(
      data_,scales,scale,num,true,false,
      matB,m,(char*)&dummy,i,j,k,w,h,d);
  float top = (a1 - b1);
  float bot = (b2 - a2 + a1 - b1);
  //degenerate cases
  if (bot == 0.) return;
  edge->isCut_eval |= CleaverCUDA::kIsCut;
  matA = matA % kMaxMaterials;
  matB = matB % kMaxMaterials;
  unsigned char minMat = min(matA,matB);
  unsigned char maxMat = max(matA,matB);
  unsigned char adder = 0;
  switch(minMat) {
    case 1: adder = 4; break;
    case 2: adder = 7; break;
    case 3: adder = 9; break;
    case 4: adder = 10; break;
    default: adder = 0; break;
  }
  unsigned char mm = (maxMat - 1 + adder)%16;
  edge->isCut_eval |= (mm << CleaverCUDA::kMaterial);
  float t = min(max(top/bot,0.f),1.f);
  float edge_verts[2][3];
  CleaverCUDA::GetEdgeVerticesCUDA(num,i,j,k,scale,edge_verts);
  for (size_t x = 0; x < 3; x++)
    edge->cut_loc[x] = (1. - t) * edge_verts[0][x] + t * edge_verts[1][x];
}

/**
 * This function determines all of the inner edge cuts.
 * @param input_device_memory Pointer to the material data.
 * @param device_scales The pointer to the scales to use.
 * @param device_scale The pointer to the scale to use.
 * @param w The input data width.
 * @param h The input data height.
 * @param d The input data depth.
 * @param m The input data number of materials.
 * @param i The x location of the start of the chunk.
 * @param j The y location of the start of the chunk.
 * @param k The z location of the start of the chunk.
 * @param endi The x location of the end of the chunk.
 * @param endj The y location of the end of the chunk.
 * @param endk The z location of the end of the chunk.
 * @param output_device_memory The pointer to the chunk of results.
 * @param which Which set of edges we're working on (inner, dual, or axis).
 */
__global__
void FindEdgeCutsCUDA(
    void* input_device_memory,
    void* device_scales,
    void* device_scale,
    size_t w, size_t h, size_t d, size_t m,
    size_t i, size_t j, size_t k,
    size_t endi, size_t endj, size_t endk,
    void* output_device_memory,
    char which) {
  //cast to proper data
  float* data_ = (float*)input_device_memory;
  float* scales = (float*)device_scales;
  float* scale = (float*)device_scale;
  Edge* edges = (Edge*)output_device_memory;
  size_t grid_max = gridDim.x * gridDim.y * gridDim.z;

  size_t idx =
      blockIdx.x       +
      blockIdx.y * gridDim.x  +
      blockIdx.z * gridDim.x * gridDim.y  +
      threadIdx.x  * grid_max  +
      threadIdx.y  * grid_max * blockDim.x +
      threadIdx.z  * grid_max * blockDim.x * blockDim.y;

  size_t num_edges = (which=='i')?8:3;

  size_t e = idx % num_edges;
  idx -= e; idx /= num_edges;
  size_t x = idx % kChunkSize;
  idx -= x; idx /= kChunkSize;
  size_t y = idx % kChunkSize;
  idx -= y; idx /= kChunkSize;
  size_t z = idx % kChunkSize;

  CleaverCUDA::edge_index edge_nums[8] =
  {DULF, DULB, DURF, DURB, DLLF, DLLB, DLRF, DLRB};
  if (which=='d') {
    edge_nums[0] = CL;
    edge_nums[1] = CD;
    edge_nums[2] = CF;
  } else if (which == 'a') {
    edge_nums[0] = LF;
    edge_nums[1] = FL;
    edge_nums[2] = LL;
  }
  //    for (size_t x = 0; x < kBlockSize; x++ )
  //      for (size_t y = 0; y < kBlockSize; y++ )
  //        for (size_t z = 0; z < kBlockSize; z++ ) {
//  for (size_t e = 0; e < num_edges; e++) {
    CleaverCUDA::edge_index edge_num = edge_nums[e];
    Edge *edge = &edges[(
        x +
        y * kChunkSize +
        z * kChunkSize * kChunkSize) * num_edges + e];
    //first clear the data
    edge->isCut_eval = 0;
    edge->cut_loc[0] = edge->cut_loc[1] = edge->cut_loc[2] = 0.0;
    if ((x < endi - i) && (y < endj - j) &&
        (z < endk - k) && (e < num_edges)) {
      FindEdgeCutCUDA(data_,scales,scale,w,h,d,m,
                      i+x,j+y,k+z,edge,edge_num);
    }
//  }
  //        }
}

__host__
void CallCUDAMaxes(float *all_data,
                   float * scales, float * scale,
                   size_t w, size_t h, size_t d, size_t m,
                   char * labels,
                   size_t wl, size_t hl, size_t dl,
                   void* device_pointers[3]) {
  //The chunk size will always be static
  size_t chunk_size = kChunkSize;
  //declare and allocate device memory for NRRD input data.
  size_t input_memory_size = w * h * d * (m - 1);
  void *input_device_memory = NULL;
  CudaCheckReturn(hipMalloc((void**)&input_device_memory,
                             sizeof(float) *
                             input_memory_size));
  CudaCheckReturn(hipMemcpy(input_device_memory, all_data,
                             sizeof(float) *
                             input_memory_size,
                             hipMemcpyHostToDevice));

  //declare and allocate memory for the output of the device.
  size_t output_memory_size = chunk_size * chunk_size * chunk_size;
  void *output_device_memory = NULL;
  CudaCheckReturn(hipMalloc((void**)&output_device_memory,
                             sizeof(char) *
                             output_memory_size));
  // allocate, set up , and copy scales.
  void *device_scales = NULL;
  CudaCheckReturn(hipMalloc((void**)&device_scales,
                             sizeof(float) * (m - 1) * 3));
  CudaCheckReturn(hipMemcpy(device_scales, scales,
                             sizeof(float) * (m - 1) * 3,
                             hipMemcpyHostToDevice));
  void *device_scale = NULL;
  CudaCheckReturn(hipMalloc((void**)&device_scale,
                             sizeof(float) * 3));
  CudaCheckReturn(hipMemcpy(device_scale, scale,
                             sizeof(float) * 3,
                             hipMemcpyHostToDevice));
  //set up blocks and grid
  dim3 dimBlock(kThreadSize, kThreadSize, kThreadSize);
  dim3 dimGrid(kBlockSize, kBlockSize, kBlockSize);
  //for each block
  for (size_t i = 0;; i+=chunk_size) {
    size_t endi = i + chunk_size;
    if (i >= wl) break;
    if (endi > wl)
      endi = wl;
    if (endi - 1 == i) break;
    for (size_t j = 0;; j+=chunk_size) {
      size_t endj = j + chunk_size;
      if (j >= hl) break;
      if (endj > hl)
        endj = hl;
      if (endj - 1 == j) break;
      for (size_t k = 0;; k+=chunk_size) {
        size_t endk = k + chunk_size;
        if (k >= dl) break;
        if (endk > dl)
          endk = dl;
        if (endk - 1 == k) break;
        //call the kernel
        FindMaxesCUDA<<<dimGrid,dimBlock>>>(
            input_device_memory, w, h, d, m,
            output_device_memory, i, j, k, endi, endj, endk,
            device_scales, device_scale);
        CudaCheckReturn(hipDeviceSynchronize());
        //copy the block results back
        char lbls[output_memory_size];
        //        FindMaxesCUDA(
        //            all_data, w, h, d, m,
        //            lbls, i, j, k, endi, endj, endk,
        //            scales, scale);
        CudaCheckReturn(hipMemcpy(lbls,output_device_memory,
                                   sizeof(char) *
                                   output_memory_size ,
                                   hipMemcpyDeviceToHost));
        for (size_t ii = 0; ii < endi - i; ii++)
          for (size_t jj = 0; jj < endj - j; jj++)
            for (size_t kk = 0; kk < endk - k; kk++) {
              labels[(i + ii) + (j + jj)*wl + (k + kk)*wl*hl] =
                  lbls[ii + jj * chunk_size + kk * chunk_size * chunk_size];
            }
      }
    }
  }
  // Free no longer used GPU memory
  CudaCheckReturn(hipFree(output_device_memory));
  device_pointers[0] = input_device_memory;
  device_pointers[1] = device_scales;
  device_pointers[2] = device_scale;
}

__host__
size_t CallCUDACuts(void* data,
                    void* scales, void* scale,
                    size_t w, size_t h, size_t d, size_t m,
                    size_t wl, size_t hl, size_t dl,
                    CleaverCUDA::Edge* inner_edges,
                    CleaverCUDA::Edge* dual_edges,
                    CleaverCUDA::Edge* axis_edges,
                    bool* cut_cells) {
  //The chunk size will always be static
  size_t chunk = kChunkSize;
  //declare and allocate output memory
  size_t output_memory_size = (chunk + 1) * (chunk + 1) * (chunk + 1) * 8;
  void *output_device_memory = NULL;
  CudaCheckReturn(hipMalloc((void**)&output_device_memory,
                             sizeof(Edge) *
                             output_memory_size));
  //the number of cuts found
  size_t count = 0, max_cell = wl*hl*dl;
  //set up blocks and grid
  dim3 dimBlock(kThreadSize, kThreadSize,kThreadSize);
  dim3 dimGrid8(kBlockSize*2, kBlockSize*2,kBlockSize*2);
  dim3 dimGrid3(kBlockSize*3, kBlockSize,kBlockSize);
  CleaverCUDA::Edge *inner_edges_output =
      new CleaverCUDA::Edge[output_memory_size];
  CleaverCUDA::Edge *dual_edges_output =
      new CleaverCUDA::Edge[output_memory_size];
  CleaverCUDA::Edge *axis_edges_output =
      new CleaverCUDA::Edge[output_memory_size];
  for (size_t t = 0; t < output_memory_size; t++) {
    inner_edges_output[t].isCut_eval = 0;
    dual_edges_output[t].isCut_eval = 0;
    axis_edges_output[t].isCut_eval = 0;
    inner_edges_output[t].cut_loc[0] =
        inner_edges_output[t].cut_loc[1] =
            inner_edges_output[t].cut_loc[2] = 0.0;
    dual_edges_output[t].cut_loc[0] =
        dual_edges_output[t].cut_loc[1] =
            dual_edges_output[t].cut_loc[2] = 0.0;
    axis_edges_output[t].cut_loc[0] =
        axis_edges_output[t].cut_loc[1] =
            axis_edges_output[t].cut_loc[2] = 0.0;
  }
  //for each block
  for (size_t i = 0;; i+=chunk) {
    size_t endi = i + chunk;
    if (i >= wl) break;
    if (endi > wl)
      endi = wl;
    if (endi - 1 == i) break;
    for (size_t j = 0;; j+=chunk) {
      size_t endj = j + chunk;
      if (j >= hl) break;
      if (endj > hl)
        endj = hl;
      if (endj - 1 == j) break;
      for (size_t k = 0;; k+=chunk) {
        size_t endk = k + chunk;
        if (k >= dl) break;
        if (endk > dl)
          endk = dl;
        if (endk - 1 == k) break;
        //call the inner edge kernel
        FindEdgeCutsCUDA<<<dimGrid8,dimBlock>>>(
            data,
            scales,
            scale,
            w, h, d, m,
            i, j, k, endi, endj, endk,
            output_device_memory,'i');
        //                FindEdgeCutsCUDA(
        //                    input_device_memoryV,
        //                    device_scalesV,
        //                    device_scaleV,
        //                    w, h, d, m,
        //                    i, j, k, endi, endj, endk,
        //                    inner_edges_output,'i');
        CudaCheckReturn(hipDeviceSynchronize());
        //copy the block results back
        CudaCheckReturn(hipMemcpy(inner_edges_output,
                                   output_device_memory,
                                   sizeof(Edge) *
                                   output_memory_size ,
                                   hipMemcpyDeviceToHost));
        //call the dual edge kernel
        FindEdgeCutsCUDA<<<dimGrid3,dimBlock>>>(
            data,
            scales,
            scale,
            w, h, d, m,
            i, j, k, endi, endj, endk,
            output_device_memory,'d');
        //                FindEdgeCutsCUDA(
        //                    input_device_memoryV,
        //                    device_scalesV,
        //                    device_scaleV,
        //                    w, h, d, m,
        //                    i, j, k, endi, endj, endk,
        //                    dual_edges_output,'d');
        CudaCheckReturn(hipDeviceSynchronize());
        //copy the block results back
        CudaCheckReturn(hipMemcpy(dual_edges_output,
                                   output_device_memory,
                                   sizeof(Edge) *
                                   output_memory_size ,
                                   hipMemcpyDeviceToHost));
        //call the axis edge kernel
        FindEdgeCutsCUDA<<<dimGrid3,dimBlock>>>(
            data,
            scales,
            scale,
            w, h, d, m,
            i, j, k, endi, endj, endk,
            output_device_memory,'a');
        //                FindEdgeCutsCUDA(
        //                    input_device_memoryV,
        //                    device_scalesV,
        //                    device_scaleV,
        //                    w, h, d, m,
        //                    i, j, k, endi, endj, endk,
        //                    axis_edges_output,'a');
        CudaCheckReturn(hipDeviceSynchronize());
        //copy the block results back
        CudaCheckReturn(hipMemcpy(axis_edges_output,
                                   output_device_memory,
                                   sizeof(Edge) *
                                   output_memory_size ,
                                   hipMemcpyDeviceToHost));
        CleaverCUDA::Edge* tmp;
        for (size_t ii = 0; ii < endi - i; ii++)
          for (size_t jj = 0; jj < endj - j; jj++)
            for (size_t kk = 0; kk < endk - k; kk++) {
              size_t cellIdx = i + ii + (j + jj)*wl + (k + kk)*wl*hl;
              size_t hostIdx = cellIdx + wl*hl;
              size_t deviceIdx = ii + jj * chunk + kk * chunk * chunk;
              //count the cuts
              for(size_t ee = 0; ee < 8; ee++) {
                //count diagonal (inner) edge cuts
                tmp = &inner_edges_output[deviceIdx*8+ee];
                if (cut_cells[cellIdx]) {
                  if ((tmp->isCut_eval & CleaverCUDA::kIsCut))
                    count++;
                } else tmp->isCut_eval = 0;
                if (ee < 3) {
                  //count dual edge cuts
                  tmp = &dual_edges_output[deviceIdx*3+ee];
                  bool include = cut_cells[cellIdx];
                  if (ee == 0 && cellIdx - 1 < max_cell)
                    include |= cut_cells[cellIdx - 1];
                  else if (ee == 1 && cellIdx - wl < max_cell)
                    include |= cut_cells[cellIdx - wl];
                  else if (ee  == 2 && cellIdx - wl*hl < max_cell)
                    include |= cut_cells[cellIdx - wl*hl];
                  //only include cut cells
                  if (!include)
                    tmp->isCut_eval = 0;
                  else if ((tmp->isCut_eval & CleaverCUDA::kIsCut))
                    count++;
                  //count axis edge cuts
                  tmp = &axis_edges_output[deviceIdx*3+ee];
                  include = cut_cells[cellIdx];
                  if (ee == 0) {
                    if(cellIdx - wl*hl < max_cell)
                      include |= cut_cells[cellIdx - wl*hl];
                    if(cellIdx - wl*hl - wl < max_cell)
                      include |= cut_cells[cellIdx - wl*hl -wl];
                    if(cellIdx - wl < max_cell)
                      include |= cut_cells[cellIdx - wl];
                  }
                  else if (ee == 1) {
                    if(cellIdx - wl*hl < max_cell)
                      include |= cut_cells[cellIdx - wl*hl];
                    if(cellIdx - wl*hl - 1 < max_cell)
                      include |= cut_cells[cellIdx - wl*hl - 1];
                    if(cellIdx - 1 < max_cell)
                      include |= cut_cells[cellIdx - 1];
                  }
                  else if (ee == 2) {
                    if(cellIdx - 1 < max_cell)
                      include |= cut_cells[cellIdx - 1];
                    if(cellIdx - 1 - wl < max_cell)
                      include |= cut_cells[cellIdx - 1 - wl];
                    if(cellIdx - wl < max_cell)
                      include |= cut_cells[cellIdx - wl];
                  }
                  //only include cut cells
                  if (!include)
                    tmp->isCut_eval = 0;
                  else if ((tmp->isCut_eval & CleaverCUDA::kIsCut))
                    count++;
                }
              }
              //inner edges copy to CPU
              memcpy(&inner_edges[hostIdx*8],
                     &inner_edges_output[deviceIdx*8], 8*sizeof(Edge));
              //dual edges copy to CPU
              memcpy(&dual_edges[hostIdx*3],
                     &dual_edges_output[deviceIdx*3], 3*sizeof(Edge));
              //axis edges copy to CPU
              memcpy(&axis_edges[hostIdx*3],
                     &axis_edges_output[deviceIdx*3], 3*sizeof(Edge));
            }
      }
    }
  }
  delete[] axis_edges_output;
  delete[] inner_edges_output;
  delete[] dual_edges_output;
  // Free GPU and reset
  CudaCheckReturn(hipFree(data));
  CudaCheckReturn(hipFree(scales));
  CudaCheckReturn(hipFree(scale));
  CudaCheckReturn(hipFree(output_device_memory));
  CudaCheckReturn(hipDeviceReset());
  return count;
}
}
